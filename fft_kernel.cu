/*****************************************************************************
 Implementation of Fast Fourier Transformation on Finite Elements
 *****************************************************************************
 * @author     Marius van der Wijden
 * @copyright  currently unlicensed, see README.md
 *****************************************************************************/


#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include <chrono>
#include <omp.h>
typedef std::chrono::high_resolution_clock Clock;


#define LOG_NUM_THREADS 11 
#define NUM_THREADS 2048
#define MULTICORE true
#define CONSTRAINTS 4194304
#define LOG_CONSTRAINTS 22

#define CUDA_CALL( call )               \
{                                       \
cudaError_t result = call;              \
if ( hipSuccess != result )            \
    std::cerr << "CUDA error " << result << " in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString( result ) << " (" << #call << ")" << std::endl;  \
}

__device__ __forceinline__
size_t bitreverse(size_t n, const size_t l)
{
    size_t r = 0;
    for (size_t k = 0; k < l; ++k)
    {
        r = (r << 1) | (n & 1);
        n >>= 1;
    }
    return r;
}

size_t bitreverse_host(size_t n, const size_t l)
{
    size_t r = 0;
    for (size_t k = 0; k < l; ++k)
    {
        r = (r << 1) | (n & 1);
        n >>= 1;
    }
    return r;
}
template<typename FieldT> 
__device__ __constant__ FieldT omega;
template<typename FieldT> 
__device__ __constant__ FieldT one;
template<typename FieldT>
__device__ FieldT field[CONSTRAINTS];

template<typename FieldT>  __global__ void cuda_fft()
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t log_m = LOG_CONSTRAINTS;
    const size_t length = CONSTRAINTS;
    const size_t block_length = 1ul << (log_m - LOG_NUM_THREADS) ;
    const size_t startidx = idx * block_length;
    assert (CONSTRAINTS == 1ul<<log_m);
    if(startidx > length)
        return;
    FieldT a [block_length];
    memset(a, block_length,  0); //TODO change to zero element
    FieldT omega_j = omega<FieldT>^idx;
    FieldT omega_step = omega<FieldT>^(idx<<(log_m - LOG_NUM_THREADS));
    
    FieldT elt = one<FieldT>;
    for (size_t i = 0; i < 1ul<<(log_m - LOG_NUM_THREADS); ++i)
    {
        for (size_t s = 0; s < NUM_THREADS; ++s)
        {
            // invariant: elt is omega^(j*idx)
            //size_t mod = (1u << log_m); //mod guaranteed to be 2^n
	        size_t id = (i + (s<<(log_m - LOG_NUM_THREADS))) % (1u << log_m);
	        if(id > length)
	            continue;
            //size_t id = (i + (s<<(log_m - LOG_NUM_THREADS))) & (mod - 1);
		a[i] += field<FieldT>[id] * elt;
                elt *= omega_step;
        }
        elt *= omega_j;
    }

    FieldT omega_num_cpus = omega<FieldT> ^ NUM_THREADS;
    
    size_t n = block_length, logn = log2f(n);
    assert (n == (1u << logn));

    /* swapping in place (from Storer's book) */
    for (size_t k = 0; k < n; ++k)
    {
        const size_t rk = bitreverse(k, logn);
        if (k < rk)
        {
            FieldT tmp = a[k];
            a[k] = a[rk];
            a[rk] = tmp;
        }
    }

    size_t m = 1; // invariant: m = 2^{s-1}
    for (size_t s = 1; s <= logn; ++s)
    {
        // w_m is 2^s-th root of unity now
        const FieldT w_m = omega_num_cpus^(n/(2*m));

        for (size_t k = 0; k < n; k += 2*m)
        {
            // FieldT * w = (FieldT *) malloc (sizeof(FieldT));
            // memcpy(w, one, sizeof(FieldT));
            FieldT w = one<FieldT>;
            for (size_t j = 0; j < m; ++j)
            {
                const FieldT t = w * a[k+j+m];
                a[k+j+m] = a[k+j] - t;
                a[k+j] += t;
                w *= w_m;
            }
        }
        m *= 2;
    }
    __syncthreads();
    for (size_t j = 0; j < 1ul<<(log_m - LOG_NUM_THREADS); ++j)
    {
        if(((j << LOG_NUM_THREADS) + idx) < length)
	       field<FieldT>[(j<<LOG_NUM_THREADS) + idx] = a[j];
    }
}

template<typename FieldT> void best_fft
    (std::vector<FieldT> &a, const FieldT &omg, const FieldT &oneElem)
    {
	FieldT* fld;
	CUDA_CALL (hipGetSymbolAddress((void **)&fld, HIP_SYMBOL(field<FieldT>)));
        CUDA_CALL( hipMemcpy(fld, &a[0], sizeof(FieldT) * a.size(), hipMemcpyHostToDevice);)
        
        hipMemcpyToSymbol(HIP_SYMBOL(omega<FieldT>), &omg, sizeof(FieldT), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(one<FieldT>), &oneElem, sizeof(FieldT), 0, hipMemcpyHostToDevice);
	
	int blocks = NUM_THREADS/1024 > 0? NUM_THREADS/1024 : 1;
	int threads = NUM_THREADS > 1024 ? 1024 : NUM_THREADS; 
        printf("blocks %d, threads %d \n",blocks,threads);
	cuda_fft<FieldT> <<<blocks,threads>>>();
        
	hipError_t error = hipGetLastError();
  	if(error != hipSuccess)
  	{
    		// print the CUDA error message and exit
    		printf("CUDA error: %s\n", hipGetErrorString(error));
    		exit(-1);
  	}
	CUDA_CALL( hipDeviceSynchronize();)

        FieldT * result = (FieldT*) malloc (sizeof(FieldT) * a.size());	
        hipMemcpy(result, fld, sizeof(FieldT) * a.size(), hipMemcpyDeviceToHost);
        a.assign(result, result + a.size());
        CUDA_CALL( hipDeviceSynchronize();)
    }

template<typename FieldT>
void _basic_serial_radix2_FFT(std::vector<FieldT> &a, const FieldT omega, const FieldT one)
{
    const size_t n = a.size(), logn = log2(n);
    
    /* swapping in place (from Storer's book) */
    for (size_t k = 0; k < n; ++k)
    {
        const size_t rk = bitreverse_host(k, logn);
        if (k < rk)
            std::swap(a[k], a[rk]);
    }

    size_t m = 1; // invariant: m = 2^{s-1}
    for (size_t s = 1; s <= logn; ++s)
    {
        // w_m is 2^s-th root of unity now
        const FieldT w_m = omega^(n/(2*m));

        asm volatile  ("/* pre-inner */");
        for (size_t k = 0; k < n; k += 2*m)
        {
            FieldT w = one;
            for (size_t j = 0; j < m; ++j)
            {
                const FieldT t = w * a[k+j+m];
                a[k+j+m] = a[k+j] - t;
                a[k+j] += t;
                w *= w_m;
            }
        }
        asm volatile ("/* post-inner */");
        m *= 2;
    }
}

template<typename FieldT>
void _basic_parallel_radix2_FFT_inner(std::vector<FieldT> &a, const FieldT omega, const size_t log_cpus, const FieldT one)
{
    const size_t num_cpus = 1ul<<log_cpus;

    const size_t m = a.size();
    const size_t log_m = log2(m);
   
    if (log_m < log_cpus)
    {
        _basic_serial_radix2_FFT(a, omega, one);
        return;
    }

    std::vector<std::vector<FieldT> > tmp(num_cpus);
    for (size_t j = 0; j < num_cpus; ++j)
    {
        tmp[j].resize(1ul<<(log_m-log_cpus), 0);
    }

    #pragma omp parallel for
    for (size_t j = 0; j < num_cpus; ++j)
    {
        const FieldT omega_j = omega^j;
        const FieldT omega_step = omega^(j<<(log_m - log_cpus));
	
	    //printf("omega_host: %d %d \n", omega_j, omega_step);
        FieldT elt = one;
        for (size_t i = 0; i < 1ul<<(log_m - log_cpus); ++i)
        {
            for (size_t s = 0; s < num_cpus; ++s)
            {
                // invariant: elt is omega^(j*idx)
                const size_t idx = (i + (s<<(log_m - log_cpus))) % (1u << log_m);
                tmp[j][i] += a[idx] * elt;
                elt *= omega_step;
            }
            elt *= omega_j;
        }
    }
    printf("host: %d \n ", tmp[0][0]);
    const FieldT omega_num_cpus = omega^num_cpus;

    #pragma omp parallel for
    for (size_t j = 0; j < num_cpus; ++j)
    {
        _basic_serial_radix2_FFT(tmp[j], omega_num_cpus, one);
    }


    #pragma omp parallel for
    for (size_t i = 0; i < num_cpus; ++i)
    {
        for (size_t j = 0; j < 1ul<<(log_m - log_cpus); ++j)
        {
            // now: i = idx >> (log_m - log_cpus) and j = idx % (1u << (log_m - log_cpus)), for idx = ((i<<(log_m-log_cpus))+j) % (1u << log_m)
            a[(j<<log_cpus) + i] = tmp[i][j];
        }
    }
}

int main(void) {

   // size_t size = 268435456;
    size_t size = CONSTRAINTS;
    //size_t size = 65536;
    int * array = (int*) malloc(size * sizeof(int));
    memset(array, 0x1234, size * sizeof(int));
    std::vector<int> v1(array, array+size);
    std::vector<int> v2 = v1;

   // printf("max_threads: %d \n", omp_get_max_threads());
    omp_set_num_threads( 8 );

    {
        {
            auto t1 = Clock::now();
            best_fft<int>(v1, 5678, 1);
            auto t2 = Clock::now();
            printf("Device FFT took %lld \n",
                std::chrono::duration_cast<
                std::chrono::milliseconds>(t2 - t1).count());
        }
        
        {
            auto t1 = Clock::now();
            _basic_parallel_radix2_FFT_inner<int> (v2, 5678, LOG_NUM_THREADS, 1);
            auto t2 = Clock::now();
            printf("Host FFT took %lld \n",
                std::chrono::duration_cast<
                std::chrono::milliseconds>(t2 - t1).count());
        }
        
        
       // _basic_parallel_radix2_FFT_inner<int> (v1, 5678, 5, 1);
    }
    

    for(int j = 0; j < size; j++) {
        //printf("%d ", v1[j]);
    }
    printf("####################################\n");
    for(int j = 0; j < size; j++) {
  //	    printf("%d ", v2[j]);
    }
    assert(v1 == v2);
    printf("\nDONE\n");
    return 0;
}

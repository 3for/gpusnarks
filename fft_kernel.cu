#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include <chrono>
#include <omp.h>
typedef std::chrono::high_resolution_clock Clock;


#define LOG_NUM_THREADS 3
#define NUM_THREADS 8
#define MULTICORE true
#define CONSTRAINTS 4194304
#define LOG_CONSTRAINTS 22

#define CUDA_CALL( call )               \
{                                       \
cudaError_t result = call;              \
if ( hipSuccess != result )            \
    std::cerr << "CUDA error " << result << " in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString( result ) << " (" << #call << ")" << std::endl;  \
}

__device__ __forceinline__
size_t bitreverse(size_t n, const size_t l)
{
    size_t r = 0;
    for (size_t k = 0; k < l; ++k)
    {
        r = (r << 1) | (n & 1);
        n >>= 1;
    }
    return r;
}

size_t bitreverse_host(size_t n, const size_t l)
{
    size_t r = 0;
    for (size_t k = 0; k < l; ++k)
    {
        r = (r << 1) | (n & 1);
        n >>= 1;
    }
    return r;
}
template<typename FieldT> 
__device__ __constant__ FieldT omega;
template<typename FieldT> 
__device__ __constant__ FieldT one;

template<typename FieldT>  __global__ void cuda_fft(
    FieldT *field, size_t const length)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d ",omega<FieldT>);
    const size_t log_m = LOG_CONSTRAINTS;
    
    const size_t block_length = 1ul << (log_m - LOG_NUM_THREADS) ;
    const size_t startidx = idx * block_length;
    assert (length == 1ul<<log_m);
    if(startidx > length)
        return;
    FieldT a[block_length];
	    //= (FieldT*)malloc(block_length * sizeof(FieldT));
    memset(a, block_length,  0); //TODO change to zero element
    FieldT omega_j = omega<FieldT>^idx;
    FieldT omega_step = omega<FieldT>^(idx<<(log_m - LOG_NUM_THREADS));
    
    FieldT elt = one<FieldT>;
    for (size_t i = 0; i < 1ul<<(log_m - LOG_NUM_THREADS); ++i)
    {
        for (size_t s = 0; s < NUM_THREADS; ++s)
        {
            // invariant: elt is omega^(j*idx)
            //size_t mod = (1u << log_m); //mod guaranteed to be 2^n
	        size_t id = (i + (s<<(log_m - LOG_NUM_THREADS))) % (1u << log_m);
	        if(id > length)
	            continue;
            //size_t id = (i + (s<<(log_m - LOG_NUM_THREADS))) & (mod - 1);
	    a[i] += field[id] * elt;
            elt *= omega_step;
        }
        elt *= omega_j;
    }

    FieldT omega_num_cpus = omega<FieldT> ^ NUM_THREADS;
    
    size_t n = block_length, logn = log2f(n);
    assert (n == (1u << logn));

    /* swapping in place (from Storer's book) */
    for (size_t k = 0; k < n; ++k)
    {
        const size_t rk = bitreverse(k, logn);
        if (k < rk)
        {
            FieldT tmp = a[k];
            a[k] = a[rk];
            a[rk] = tmp;
        }
    }

    size_t m = 1; // invariant: m = 2^{s-1}
    for (size_t s = 1; s <= logn; ++s)
    {
        // w_m is 2^s-th root of unity now
        const FieldT w_m = omega_num_cpus^(n/(2*m));

        for (size_t k = 0; k < n; k += 2*m)
        {
            // FieldT * w = (FieldT *) malloc (sizeof(FieldT));
            // memcpy(w, one, sizeof(FieldT));
            FieldT w = one<FieldT>;
            for (size_t j = 0; j < m; ++j)
            {
                const FieldT t = w * a[k+j+m];
                a[k+j+m] = a[k+j] - t;
                a[k+j] += t;
                w *= w_m;
            }
        }
        m *= 2;
    }

    for (size_t j = 0; j < 1ul<<(log_m - LOG_NUM_THREADS); ++j)
    {
        // now: i = idx >> (log_m - log_cpus) and j = idx % (1u << (log_m - log_cpus)), for idx = ((i<<(log_m-log_cpus))+j) % (1u << log_m)
        if(((j << LOG_NUM_THREADS) + idx) < length)
	       field[(j<<LOG_NUM_THREADS) + idx] = a[j];
    }
    free(a);
}

template<typename FieldT> void best_fft
    (std::vector<FieldT> &a, const FieldT &omg, const FieldT &oneElem)
    {
        FieldT * array;
        CUDA_CALL( hipMalloc((void**) &array, sizeof(FieldT) * a.size());)
        CUDA_CALL( hipMemcpy(array, &a[0], sizeof(FieldT) * a.size(), hipMemcpyHostToDevice);)
        

        CUDA_CALL (hipMemcpyToSymbol(HIP_SYMBOL(omega<FieldT>), &omg, sizeof(FieldT), 0, hipMemcpyHostToDevice);)
        CUDA_CALL (hipMemcpyToSymbol(HIP_SYMBOL(one<FieldT>), &oneElem, sizeof(FieldT), 0, hipMemcpyHostToDevice);)

	    int blocks = NUM_THREADS/1024 + 1;
	    int threads = NUM_THREADS > 1024 ? 1024 : NUM_THREADS; 
        cuda_fft<FieldT><<<blocks,threads>>>(array, a.size());
        CUDA_CALL( hipDeviceSynchronize();)

        FieldT * result = (FieldT*) malloc (sizeof(FieldT) * a.size());	
        hipMemcpy(result, array, sizeof(FieldT) * a.size(), hipMemcpyDeviceToHost);
        a.assign(result, result + a.size());
        CUDA_CALL( hipDeviceSynchronize();)
 //       printf("%d tick", result[3]);
    }

template<typename FieldT>
void _basic_serial_radix2_FFT(std::vector<FieldT> &a, const FieldT omega, const FieldT one)
{
    const size_t n = a.size(), logn = log2(n);
    
    /* swapping in place (from Storer's book) */
    for (size_t k = 0; k < n; ++k)
    {
        const size_t rk = bitreverse_host(k, logn);
        if (k < rk)
            std::swap(a[k], a[rk]);
    }

    size_t m = 1; // invariant: m = 2^{s-1}
    for (size_t s = 1; s <= logn; ++s)
    {
        // w_m is 2^s-th root of unity now
        const FieldT w_m = omega^(n/(2*m));

        asm volatile  ("/* pre-inner */");
        for (size_t k = 0; k < n; k += 2*m)
        {
            FieldT w = one;
            for (size_t j = 0; j < m; ++j)
            {
                const FieldT t = w * a[k+j+m];
                a[k+j+m] = a[k+j] - t;
                a[k+j] += t;
                w *= w_m;
            }
        }
        asm volatile ("/* post-inner */");
        m *= 2;
    }
}

template<typename FieldT>
void _basic_parallel_radix2_FFT_inner(std::vector<FieldT> &a, const FieldT omega, const size_t log_cpus, const FieldT one)
{
    const size_t num_cpus = 1ul<<log_cpus;

    const size_t m = a.size();
    const size_t log_m = log2(m);
   
    if (log_m < log_cpus)
    {
        _basic_serial_radix2_FFT(a, omega, one);
        return;
    }

    std::vector<std::vector<FieldT> > tmp(num_cpus);
    for (size_t j = 0; j < num_cpus; ++j)
    {
        tmp[j].resize(1ul<<(log_m-log_cpus), 0);
    }

    #pragma omp parallel for
    for (size_t j = 0; j < num_cpus; ++j)
    {
        const FieldT omega_j = omega^j;
        const FieldT omega_step = omega^(j<<(log_m - log_cpus));
	
	    //printf("omega_host: %d %d \n", omega_j, omega_step);
        FieldT elt = one;
        for (size_t i = 0; i < 1ul<<(log_m - log_cpus); ++i)
        {
            for (size_t s = 0; s < num_cpus; ++s)
            {
                // invariant: elt is omega^(j*idx)
                const size_t idx = (i + (s<<(log_m - log_cpus))) % (1u << log_m);
                tmp[j][i] += a[idx] * elt;
                elt *= omega_step;
            }
            elt *= omega_j;
        }
    }
    printf("host: %d \n ", tmp[0][0]);
    const FieldT omega_num_cpus = omega^num_cpus;

    #pragma omp parallel for
    for (size_t j = 0; j < num_cpus; ++j)
    {
        _basic_serial_radix2_FFT(tmp[j], omega_num_cpus, one);
    }


    #pragma omp parallel for
    for (size_t i = 0; i < num_cpus; ++i)
    {
        for (size_t j = 0; j < 1ul<<(log_m - log_cpus); ++j)
        {
            // now: i = idx >> (log_m - log_cpus) and j = idx % (1u << (log_m - log_cpus)), for idx = ((i<<(log_m-log_cpus))+j) % (1u << log_m)
            a[(j<<log_cpus) + i] = tmp[i][j];
        }
    }
}

int main(void) {

   // size_t size = 268435456;
    size_t size = CONSTRAINTS;
    //size_t size = 65536;
    int * array = (int*) malloc(size * sizeof(int));
    memset(array, 0x1234, size * sizeof(int));
    std::vector<int> v1(array, array+size);
    std::vector<int> v2 = v1;

   // printf("max_threads: %d \n", omp_get_max_threads());
    omp_set_num_threads( 8 );

    {
        {
            auto t1 = Clock::now();
            best_fft<int>(v1, 5678, 1);
            auto t2 = Clock::now();
            printf("Device FFT took %lld \n",
                std::chrono::duration_cast<
                std::chrono::milliseconds>(t2 - t1).count());
        }
        
        {
            auto t1 = Clock::now();
            _basic_parallel_radix2_FFT_inner<int> (v2, 5678, LOG_NUM_THREADS, 1);
            auto t2 = Clock::now();
            printf("Host FFT took %lld \n",
                std::chrono::duration_cast<
                std::chrono::milliseconds>(t2 - t1).count());
        }
        
        
       // _basic_parallel_radix2_FFT_inner<int> (v1, 5678, 5, 1);
    }
    

    for(int j = 0; j < size; j++) {
        //printf("%d ", v1[j]);
    }
    printf("####################################\n");
    for(int j = 0; j < size; j++) {
  //	    printf("%d ", v2[j]);
    }
    assert(v1 == v2);
    printf("\nDONE\n");
    return 0;
}
